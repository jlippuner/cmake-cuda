#include "hip/hip_runtime.h"
#include "example.cuh"

#include <cstdio>
#include <stdexcept>

#include <hip/hip_runtime.h>

#define CUCHK(cmd) {                                                            \
  auto errorCode = (cmd);                                                       \
  if (errorCode != hipSuccess) {                                               \
    char buf[1024];                                                             \
    sprintf(buf, "CUDA ERROR %i at %s:%i: %s", errorCode, __FILE__, __LINE__,   \
        hipGetErrorString(errorCode));                                         \
    throw std::runtime_error(buf);                                              \
  }                                                                             \
}

__global__ void kernel(int *num_threads) {
  if ((threadIdx.x == 0) && (blockIdx.x == 0)) {
    *num_threads = blockDim.x * gridDim.x;
  }
}

void HelloWorld_Device() {
  int *d_num_threads, *h_num_threads;

  // allocate memory
  h_num_threads = (int*)malloc(sizeof(int));
  CUCHK(hipMalloc(&d_num_threads, sizeof(int)));

  // run kernel
  kernel<<<256, 8>>>(d_num_threads);
  CUCHK(hipDeviceSynchronize());

  // copy result back to host
  CUCHK(hipMemcpy(h_num_threads, d_num_threads, sizeof(int), hipMemcpyDeviceToHost));

  printf("Hello world from GPU with %i threads\n", *h_num_threads);

  // free memory
  free(h_num_threads);
  CUCHK(hipFree(d_num_threads));
}
